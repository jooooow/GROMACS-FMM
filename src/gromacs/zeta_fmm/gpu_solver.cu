#include "hip/hip_runtime.h"
#include "solver.h"
#include "gromacs/math/units.h"

zeta_fmm::GPUSolver::GPUSolver(FILE* f) : Solver(f)
{
    //fprintf(fplog, "init non-para GPUSolver hjm\n");
}

zeta_fmm::GPUSolver::GPUSolver(
    FILE* f,
    std::vector<Body3>& bs,
    int P_,
    zeta_fmm::real box_r_,
    zeta_fmm::vec3r center_,
    int max_depth_,
    int images_,
    zeta_fmm::real cycle_,
    zeta_fmm::real rega_,
    int verbose_,
    int dummy_
) : Solver(f, bs, "gpuFMM"), P(P_), box_r(box_r_), center(center_), max_depth(max_depth_), images(images_), cycle(cycle_), rega(rega_), verbose(verbose_), dummy(dummy_)
{
    merged_num_body = bs.size();
    merged_bodies = bs;
    if(verbose)
    {
        printf("[verbose] bs.size = %d\n", merged_num_body);
    }
}

zeta_fmm::GPUSolver::~GPUSolver()
{
    
}

void zeta_fmm::GPUSolver::prepare_maps()
{
    hipFree(0);
    
    // build tree from bodies
    zeta_fmm::CompleteBalancedOctree tree;
    tree.build(bodies, box_r, center, max_depth);
    tree.build_reg(bodies, rega, max_depth, cycle); // caution : this step will extend the bodies(thus bodies's real size changes)

    // set near field policy
    zeta_fmm::NearFieldSelector nearfield_selector(zeta_fmm::NearFieldSelector::SelectorType::MAC, 0.4);

    // traverse tree to get interaction map
    zeta_fmm::Traverser traverser;
    traverser.traverse(tree, nearfield_selector, images, cycle); // caution : when images > 0, this function will add extra cells
    cells = traverser.get_cells();
    depth_map = traverser.get_depth_map();
    p2p_map = traverser.get_p2p_map();
    m2l_map = traverser.get_m2l_map();

    if(verbose)
    {
        printf("p2p_map.size = %ld\n", p2p_map.size());
        printf("m2l_map.size = %ld\n", m2l_map.size());
    }
}

void zeta_fmm::GPUSolver::solve()
{
    //fprintf(fplog, "GPUSolver solve hjm\n");

    // prepare interaction maps
    prepare_maps();

    // prepare body & cells
    int real_num_body = bodies.size();
    int real_num_cell = cells.size();
    
    if(verbose)
    {
        printf("merged_num_body = %d, real_num_body = %d, real_num_cell = %d\n", merged_num_body, real_num_body, real_num_cell);
    }
    
    Body3* g_bodies;
    hipMalloc(&g_bodies, real_num_body * sizeof(Body3));
    hipMemcpy(g_bodies, bodies.data(), real_num_body * sizeof(Body3), hipMemcpyHostToDevice);

    Body3* g_merged_bodies;
    hipMalloc(&g_merged_bodies, merged_num_body * sizeof(Body3));
    hipMemcpy(g_merged_bodies, merged_bodies.data(), merged_num_body * sizeof(Body3), hipMemcpyHostToDevice);

    Cell3* g_cells;
    hipMalloc(&g_cells, real_num_cell * sizeof(Cell3));
    hipMemcpy(g_cells, cells.data(), real_num_cell * sizeof(Cell3), hipMemcpyHostToDevice);

    // prepare pole
    int pole_size_eachcell = get_pole_size_eachcell(P);
    zeta_fmm::complex* g_Ms;
    hipMalloc(&g_Ms, real_num_cell * pole_size_eachcell * sizeof(complex));
    hipMemset(g_Ms, 0, real_num_cell * pole_size_eachcell * sizeof(complex));
    zeta_fmm::complex* g_Ls;
    hipMalloc(&g_Ls, real_num_cell * pole_size_eachcell * sizeof(complex));
    hipMemset(g_Ls, 0, real_num_cell * pole_size_eachcell * sizeof(complex));

    // find max_depth
    int real_max_depth = get_real_max_depth();
    if(verbose)
    {
        std::cout<<"real_max_depth = "<<real_max_depth<<std::endl;
    }

    // get leaf cell (P2M & L2P)
    std::vector<int> leaf_list = get_leaf_list(); // max-depth-cells
    int leaf_cell_num = leaf_list.size();
    if(verbose)
    {
        std::cout<<"leaf_cell_num = "<<leaf_cell_num<<std::endl;
    }
    int* g_leaf_cells;
    hipMalloc(&g_leaf_cells, leaf_cell_num * sizeof(int));
    hipMemcpy(g_leaf_cells, leaf_list.data(), leaf_cell_num * sizeof(int), hipMemcpyHostToDevice);

    // store sorted branch cell (M2M & L2L)
    std::vector<int> branch_cells; // cells except max-depth-cells and image-cells (sorted from max-depth to 0)
    std::vector<OffsetAndNumber> level_infos;
    int level_offset = 0;
    for(auto map = depth_map.rbegin(); map != depth_map.rend(); map++)
    {
        if(map->first == real_max_depth || map->first < 0) continue;
        std::vector<IndexAndOffset3r>& cs = map->second;
        level_infos.push_back(OffsetAndNumber(level_offset, cs.size()));
        for(size_t i = 0; i < cs.size(); i++)
        {
            branch_cells.push_back(cs[i].idx);
        }
        level_offset += cs.size();
    }
    int* g_branch_cells;
    hipMalloc(&g_branch_cells, branch_cells.size() * sizeof(int));
    hipMemcpy(g_branch_cells, branch_cells.data(), branch_cells.size() * sizeof(int), hipMemcpyHostToDevice);

    // get image cells
    std::vector<int> img_cells; // image-cells
    for(auto map = depth_map.rbegin(); map != depth_map.rend(); map++)
    {
        if(map->first < 0)
        {
            img_cells.push_back(map->second[0].idx);
        }
    }
    int img_cells_num = img_cells.size();
    if(verbose)
    {
       printf("img_cells_num = %d\n", img_cells_num); 
    }
    int* g_img_cells;
    hipMalloc(&g_img_cells, img_cells_num * sizeof(int));
    hipMemcpy(g_img_cells, img_cells.data(), img_cells_num * sizeof(int), hipMemcpyHostToDevice);

    // generate p2p matrix (P2P)
    int max_p2p_source_num = 0;
    for(auto map : p2p_map)
    {
        max_p2p_source_num = std::max(max_p2p_source_num, (int)map.second.size());
    }
    int p2p_matrix_row = p2p_map.size();
    int p2p_matrix_col = max_p2p_source_num + 2;
    int* c_p2p_matrix = new int[p2p_matrix_row * p2p_matrix_col];
    Offset3rPadding* c_p2p_offset_matrix = new Offset3rPadding[p2p_matrix_row * (p2p_matrix_col - 2)];
    int i = 0;
    for(auto map : p2p_map)
    {
        c_p2p_matrix[i * p2p_matrix_col + 0] = map.first;
        c_p2p_matrix[i * p2p_matrix_col + 1] = map.second.size();
        for(size_t j = 0; j < map.second.size(); j++)
        {
            c_p2p_matrix[i * p2p_matrix_col + 2 + j] = map.second[j].idx;
            Offset3rPadding offset;
            offset.x = map.second[j].offset.x;
            offset.y = map.second[j].offset.y;
            offset.z = map.second[j].offset.z;
            c_p2p_offset_matrix[i * (p2p_matrix_col - 2) + j] = offset;
        }
        i++;
    }
    int* g_p2p_matrix;
    hipMalloc(&g_p2p_matrix, p2p_matrix_row * p2p_matrix_col * sizeof(int));
    hipMemcpy(g_p2p_matrix, c_p2p_matrix, p2p_matrix_row * p2p_matrix_col * sizeof(int),hipMemcpyHostToDevice);
    Offset3rPadding* g_p2p_offset_matrix;
    hipMalloc(&g_p2p_offset_matrix, p2p_matrix_row * (p2p_matrix_col - 2) * sizeof(Offset3rPadding));
    hipMemcpy(g_p2p_offset_matrix, c_p2p_offset_matrix, p2p_matrix_row * (p2p_matrix_col - 2) * sizeof(Offset3rPadding),hipMemcpyHostToDevice);

    // prepare m2l matrix (M2L)
    int max_m2l_source_num = 0;
    for(auto map : m2l_map)
    {
        max_m2l_source_num = std::max(max_m2l_source_num, (int)map.second.size());
    }
    int m2l_matrix_row = m2l_map.size();
    int m2l_matrix_col = max_m2l_source_num + 2;
    int* c_m2l_matrix = new int[m2l_matrix_row * m2l_matrix_col];
    Offset3rPadding* c_m2l_offset_matrix = new Offset3rPadding[m2l_matrix_row * (m2l_matrix_col - 2)];
    i = 0;
    for(auto map : m2l_map)
    {
        c_m2l_matrix[i * m2l_matrix_col + 0] = map.first;
        c_m2l_matrix[i * m2l_matrix_col + 1] = map.second.size();
        for(size_t j = 0; j < map.second.size(); j++)
        {
            c_m2l_matrix[i * m2l_matrix_col + 2 + j] = map.second[j].idx;
            Offset3rPadding offset;
            offset.x = map.second[j].offset.x;
            offset.y = map.second[j].offset.y;
            offset.z = map.second[j].offset.z;
            c_m2l_offset_matrix[i * (m2l_matrix_col - 2) + j] = offset;
        }
        i++;
    }
    int* g_m2l_matrix;
    hipMalloc(&g_m2l_matrix, m2l_matrix_row * m2l_matrix_col * sizeof(int));
    hipMemcpy(g_m2l_matrix, c_m2l_matrix, m2l_matrix_row * m2l_matrix_col * sizeof(int),hipMemcpyHostToDevice);
    Offset3rPadding* g_m2l_offset_matrix;
    hipMalloc(&g_m2l_offset_matrix, m2l_matrix_row * (m2l_matrix_col - 2) * sizeof(Offset3rPadding));
    hipMemcpy(g_m2l_offset_matrix, c_m2l_offset_matrix, m2l_matrix_row * (m2l_matrix_col - 2) * sizeof(Offset3rPadding),hipMemcpyHostToDevice);

    // solve FMM
    //TIME_BEGIN(gpuFMM_kernels);
    gpu_kernel.solve(
        P, rega,
        g_bodies, real_num_body, 
        g_merged_bodies, merged_num_body,
        g_cells, real_num_cell,
        g_Ms, g_Ls,
        g_leaf_cells, leaf_cell_num, 
        g_branch_cells, level_infos,
        g_img_cells, img_cells_num, cycle,
        g_p2p_matrix, g_p2p_offset_matrix, p2p_matrix_row, p2p_matrix_col,
        g_m2l_matrix, g_m2l_offset_matrix, m2l_matrix_row, m2l_matrix_col,
        dummy
    );
    hipDeviceSynchronize();
    //TIME_END(gpuFMM_kernels);

    // store result
    if(rega == 0)
    {
        hipMemcpy(bodies.data(), g_bodies, real_num_body * sizeof(Body3), hipMemcpyDeviceToHost);
    }
    else
    {
        hipMemcpy(merged_bodies.data(), g_merged_bodies, merged_num_body * sizeof(Body3), hipMemcpyDeviceToHost);
        bodies = merged_bodies;
    }

    // convert -E to F
    for(size_t i = 0; i < bodies.size(); i++)
    {
        zeta_fmm::Body3& b = bodies[i];
        b.p *= gmx::c_one4PiEps0;
        b.f *= -b.q * gmx::c_one4PiEps0;
    }

    // memory dellocate
    hipFree(g_bodies);
    hipFree(g_merged_bodies);
    hipFree(g_cells);
    hipFree(g_leaf_cells);
    hipFree(g_branch_cells);
    hipFree(g_img_cells);
    hipFree(g_p2p_matrix);
    hipFree(g_p2p_offset_matrix);
    hipFree(g_m2l_matrix);
    hipFree(g_m2l_offset_matrix);
    hipFree(g_Ms);
    hipFree(g_Ls);

    // check error
    CHECK_CUDA_ERR(overall);

    delete[] c_p2p_matrix;
    delete[] c_p2p_offset_matrix;
    delete[] c_m2l_matrix;
    delete[] c_m2l_offset_matrix;
}

std::vector<int> zeta_fmm::GPUSolver::get_leaf_list()
{
    std::vector<IndexAndOffset3r>& leaf_cell_list = depth_map[get_real_max_depth()];
    std::vector<int> leaf_list;
    for(size_t i = 0; i < leaf_cell_list.size(); i++)
    {
        IndexAndOffset3r cell = leaf_cell_list[i];
        leaf_list.push_back(cell.idx);
    }
    return leaf_list;
}

int zeta_fmm::GPUSolver::get_real_max_depth()
{
    int m = 0;
    for(auto map = depth_map.rbegin(); map != depth_map.rend(); map++)
    {
        m = std::max(m, map->first);
    }
    return m;
}
