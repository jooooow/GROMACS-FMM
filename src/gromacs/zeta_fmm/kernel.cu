#include "hip/hip_runtime.h"
#include "kernel.h"

#define P2P_KERNEL_TH_NUM 64

__device__ inline zeta_fmm::real reg_w(zeta_fmm::real x)
{
    return 0.25 * (2 + 3 * x - x * x * x);
}

__device__ inline zeta_fmm::real reg_dw(zeta_fmm::real x)
{
    return 0.25 * (3 - 3 * x * x);
}

__device__ inline zeta_fmm::real get_w_single(zeta_fmm::real dx, zeta_fmm::real R, zeta_fmm::real rega)
{
    zeta_fmm::real r = std::abs(dx) - R + rega;
    zeta_fmm::real t;
    if(r <= 0) t = 1;
    else if(r > 2 * rega) t = -1;
    else t = 1 - r / rega;
    return reg_w(t);
}

__device__ inline zeta_fmm::real get_dw_single(zeta_fmm::real dx, zeta_fmm::real R, zeta_fmm::real rega)
{
    zeta_fmm::real r = std::abs(dx) - R + rega;
    zeta_fmm::real dw;
    if(r > 0 && r <= 2 * rega)
    {
        dw = reg_dw(1 - r / rega) * (-1 / rega) * (dx > 0 ? 1 : -1);
    }
    else
    {
        dw = 0;
    }

    return dw;
}

__device__ inline zeta_fmm::real get_w(zeta_fmm::vec3r dx, zeta_fmm::real R, zeta_fmm::real rega)
{
    zeta_fmm::real w = 1;
    for(int d = 0; d < 3; d++)
    {
        w *= get_w_single(dx[d], R, rega);
    }
    return w;
}

__device__ inline zeta_fmm::vec3r get_dw(zeta_fmm::vec3r dx, zeta_fmm::real R, zeta_fmm::real rega)
{
    zeta_fmm::vec3r dw;
    dw[0] = get_dw_single(dx[0], R, rega) * get_w_single(dx[1], R, rega) * get_w_single(dx[2], R, rega);
    dw[1] = get_w_single(dx[0], R, rega) * get_dw_single(dx[1], R, rega) * get_w_single(dx[2], R, rega);
    dw[2] = get_w_single(dx[0], R, rega) * get_w_single(dx[1], R, rega) * get_dw_single(dx[2], R, rega);
    return dw;
}

__device__ inline zeta_fmm::vec3r add(zeta_fmm::vec3r a, zeta_fmm::Offset3rPadding b)
{
    zeta_fmm::vec3r c;
    c[0] = a[0] + b.x;
    c[1] = a[1] + b.y;
    c[2] = a[2] + b.z;
    return c;
}

__device__ int oddOrEven(int n)
{
    return (((n) & 1) == 1) ? -1 : 1;
}

__device__ int ipow2n(int n) 
{
    return (n >= 0) ? 1 : oddOrEven(n);
}

//d/d(r) -> d/d(x)
__device__ zeta_fmm::vec3r derivate_sph2cart(zeta_fmm::Sph dev_sph_cart, zeta_fmm::vec3r dev_p_sph) 
{
    zeta_fmm::real sin_theta = std::sin(dev_sph_cart.theta);
    zeta_fmm::real inv_sin_theta = sin_theta == 0 ? 0 : 1 / sin_theta;
    zeta_fmm::real inv_rho = dev_sph_cart.rho == 0 ? 0 : 1 / dev_sph_cart.rho;

    zeta_fmm::vec3r cart;
    cart[0] = sin_theta * std::cos(dev_sph_cart.phi) * dev_p_sph[0]
            + std::cos(dev_sph_cart.theta) * std::cos(dev_sph_cart.phi) * inv_rho * dev_p_sph[1]
            - std::sin(dev_sph_cart.phi) * inv_rho * inv_sin_theta * dev_p_sph[2];
    cart[1] = sin_theta * std::sin(dev_sph_cart.phi) * dev_p_sph[0]
            + std::cos(dev_sph_cart.theta) * std::sin(dev_sph_cart.phi) * inv_rho * dev_p_sph[1]
            + std::cos(dev_sph_cart.phi) * inv_rho * inv_sin_theta * dev_p_sph[2];
    cart[2] = std::cos(dev_sph_cart.theta) * dev_p_sph[0]
            - sin_theta * inv_rho * dev_p_sph[1];
    return cart;
}

__device__ void calc_Ynm(zeta_fmm::complex* Ynm, int P, zeta_fmm::real rho, zeta_fmm::real theta, zeta_fmm::real phi)
{
    zeta_fmm::real x = std::cos(theta);
    zeta_fmm::real y = std::sin(theta);
    zeta_fmm::real fact = 1;
    zeta_fmm::real pn = 1;
    zeta_fmm::real rhom = 1;
    zeta_fmm::complex ei;
    ei.rel = 0; ei.img = phi;
    ei = ei.exp();
    zeta_fmm::complex eim;
    eim.rel = 1; eim.img = 0;
    for (int m=0; m<=P; m++)
    {
        zeta_fmm::real p = pn;
        int npn = m * m + 2 * m;
        int nmn = m * m;
        Ynm[npn] = rhom * p * eim;
        Ynm[nmn] = Ynm[npn].conj();
        zeta_fmm::real p1 = p;
        p = x * (2 * m + 1) * p1;
        rhom *= rho;
        zeta_fmm::real rhon = rhom;
        for (int n=m+1; n<=P; n++) 
        {
            int npm = n * n + n + m;
            int nmm = n * n + n - m;
            rhon /= -(n + m);
            Ynm[npm] = rhon * p * eim;
            Ynm[nmm] = Ynm[npm].conj();
            zeta_fmm::real p2 = p1;
            p1 = p;
            p = (x * (2 * n + 1) * p1 - (n + m) * p2) / (n - m + 1);
            rhon *= rho;
        }
        rhom /= -(2 * m + 2) * (2 * m + 1);
        pn = -pn * fact * y;
        fact += 2;
        eim *= ei;
    } 
}

__device__ void calc_YnmD(zeta_fmm::complex* YnmD, int P, zeta_fmm::real rho, zeta_fmm::real theta, zeta_fmm::real phi)
{
    zeta_fmm::real x = std::cos(theta);
    zeta_fmm::real y = std::sin(theta);
    zeta_fmm::real invY = y == 0 ? 0 : 1 / y;
    zeta_fmm::real fact = 1;
    zeta_fmm::real pn = 1;
    zeta_fmm::real rhom = 1;
    zeta_fmm::complex ei;
    ei.rel = 0; ei.img = phi;
    ei = ei.exp();
    zeta_fmm::complex eim;
    eim.rel = 1; eim.img = 0;
    for (int m=0; m<=P; m++)
    {
        zeta_fmm::real p = pn;
        int npn = m * m + 2 * m;
        zeta_fmm::real p1 = p;
        p = x * (2 * m + 1) * p1;
        YnmD[npn] = rhom * (p - (m + 1) * x * p1) * invY * eim;
        rhom *= rho;
        zeta_fmm::real rhon = rhom;
        for (int n=m+1; n<=P; n++) 
        {
            int npm = n * n + n + m;
            rhon /= -(n + m);
            zeta_fmm::real p2 = p1;
            p1 = p;
            p = (x * (2 * n + 1) * p1 - (n + m) * p2) / (n - m + 1);
            YnmD[npm] = rhon * ((n - m + 1) * p - (n + 1) * x * p1) * invY * eim;
            rhon *= rho;
        }
        rhom /= -(2 * m + 2) * (2 * m + 1);
        pn = -pn * fact * y;
        fact += 2;
        eim *= ei;
    } 
}

__device__ void calc_Ynm2(zeta_fmm::complex* Ynm, int P, zeta_fmm::real rho, zeta_fmm::real theta, zeta_fmm::real phi)
{
    zeta_fmm::real x = std::cos(theta);                                
    zeta_fmm::real y = std::sin(theta);                                
    zeta_fmm::real fact = 1;                                           
    zeta_fmm::real pn = 1;                                             
    zeta_fmm::real invR = -1.0 / rho;                                  
    zeta_fmm::real rhom = -invR;                                       
    zeta_fmm::complex ei;
    ei.rel = 0; ei.img = phi;
    ei = ei.exp();
    zeta_fmm::complex eim;
    eim.rel = 1; eim.img = 0;                                
    for (int m=0; m<=P; m++) 
    {                                  
        zeta_fmm::real p = pn;                                           
        int npn = m * m + 2 * m;                                 
        int nmn = m * m;                                         
        Ynm[npn] = rhom * p * eim;                               
        Ynm[nmn] = Ynm[npn].conj();                          
        zeta_fmm::real p1 = p;                                           
        p = x * (2 * m + 1) * p1;                                
        rhom *= invR;                                            
        zeta_fmm::real rhon = rhom;                                      
        for (int n=m+1; n<=P; n++) {                              
            int npm = n * n + n + m;                               
            int nmm = n * n + n - m;                               
            Ynm[npm] = rhon * p * eim;                             
            Ynm[nmm] = Ynm[npm].conj();                        
            zeta_fmm::real p2 = p1;                                        
            p1 = p;                                                
            p = (x * (2 * n + 1) * p1 - (n + m) * p2) / (n - m + 1);
            rhon *= invR * (n - m + 1);                            
        }                                                        
        pn = -pn * fact * y;                                     
        fact += 2;                                               
        eim *= ei;                                               
    }                                                          
}

__global__ void p2m(zeta_fmm::Body3* bodies, zeta_fmm::Cell3* cells, int* leaf_cells, int P, zeta_fmm::complex* Ms)
{
    int cell_idx = leaf_cells[blockIdx.x];
    int pole_size_eachcell = get_pole_size_eachcell(P);
    zeta_fmm::Cell3* cell = cells + cell_idx;
    zeta_fmm::complex* M = Ms + cell_idx * pole_size_eachcell;

    __shared__ zeta_fmm::complex Ynm[512];

    for(int i = 0; i < cell->body_info.number; i++)
    {
        zeta_fmm::Body3* body = bodies + cell->body_info.offset + i;
        zeta_fmm::vec3r dx = body->loc - cell->center;
        zeta_fmm::Sph sph = dx.toSph2();
        calc_Ynm(Ynm, P, sph.rho, sph.theta, -sph.phi);
        for (int n = 0; n <= P; n++) 
        {
            for (int m = 0; m <= n; m++) 
            {
                M[n * (n + 1) / 2 + m] += body->q * Ynm[n * n + n + m];
            }
        }
    }
}

__global__ void p2m_dummy(zeta_fmm::Body3* bodies, zeta_fmm::Cell3* cells, int* leaf_cells, int P, zeta_fmm::complex* Ms)
{
    int cell_idx = leaf_cells[blockIdx.x];
    int pole_size_eachcell = get_pole_size_eachcell(P);
    zeta_fmm::Cell3* cell = cells + cell_idx;
    zeta_fmm::complex* M = Ms + cell_idx * pole_size_eachcell;

    __shared__ zeta_fmm::complex Ynm[512];

    for(int i = 0; i < cell->body_info.number; i++)
    {
        zeta_fmm::Body3* body = bodies + cell->body_info.offset + i;
        zeta_fmm::vec3r dx = body->loc - cell->center;
        zeta_fmm::Sph sph = dx.toSph2();
        calc_Ynm(Ynm, P, sph.rho, sph.theta, -sph.phi);
        int have_nan = 0;
        for (int n = 0; n <= P; n++) 
        {
            for (int m = 0; m <= n; m++) 
            {
                M[n * (n + 1) / 2 + m] = Ynm[n * n + n + m];
                if(isnan(Ynm[n * n + n + m].rel) || isnan(Ynm[n * n + n + m].rel))
                {
                    have_nan = 1;
                }
            }
        }
        if(have_nan)
        {
            M[0].rel = dx[0];
            M[0].img = dx[1];
            M[1].rel = dx[2];
            M[2].rel = sph.rho;
            M[2].img = sph.theta;
            M[3].rel = sph.phi;
            return;
        }
    }
}

__global__ void p2m_reg(zeta_fmm::Body3* bodies, zeta_fmm::Cell3* cells, int* leaf_cells, int P, zeta_fmm::complex* Ms, zeta_fmm::real rega)
{
    int cell_idx = leaf_cells[blockIdx.x];
    int pole_size_eachcell = get_pole_size_eachcell(P);
    zeta_fmm::Cell3* cell = cells + cell_idx;
    zeta_fmm::complex* M = Ms + cell_idx * pole_size_eachcell;

    __shared__ zeta_fmm::complex Ynm[512];

    for(int i = 0; i < cell->body_info.number; i++)
    {
        zeta_fmm::Body3* body = bodies + cell->body_info.offset + i;
        zeta_fmm::vec3r dx = body->loc - cell->center;
        zeta_fmm::Sph sph = dx.toSph2();
        calc_Ynm(Ynm, P, sph.rho, sph.theta, -sph.phi);
        zeta_fmm::real w = get_w(dx, cell->r, rega);
        for (int n = 0; n <= P; n++) 
        {
            for (int m = 0; m <= n; m++) 
            {
                M[n * (n + 1) / 2 + m] += w * body->q * Ynm[n * n + n + m];
            }
        }
    }
}

__global__ void m2m(zeta_fmm::Cell3* cells, int* branch_cells, int offset, int P, zeta_fmm::complex* Ms)
{
    int cell_idx = branch_cells[blockIdx.x + offset];
    int pole_size_eachcell = get_pole_size_eachcell(P);
    zeta_fmm::Cell3* cell = cells + cell_idx;
    zeta_fmm::complex* M = Ms + cell_idx * pole_size_eachcell;

    __shared__ zeta_fmm::complex Ynm[512];

    for(int i = 0; i < cell->child_info.number; i++)
    {
        int child_cell_idx = cell->child_info.offset + i;
        zeta_fmm::Cell3* child_cell = cells + child_cell_idx;
        zeta_fmm::complex* child_M = Ms + child_cell_idx * pole_size_eachcell;
        
        zeta_fmm::vec3r dx = cell->center - child_cell->center;
        zeta_fmm::Sph sph = dx.toSph2();
        calc_Ynm(Ynm, P, sph.rho, sph.theta, sph.phi);
        for(int j = 0; j <= P; j++) 
        {
            for (int k = 0; k <= j; k++) 
            {
                int jks = j * (j + 1) / 2 + k;
                zeta_fmm::complex temp_M;
                temp_M.rel = 0; temp_M.img = 0;
                for (int n=0; n<=j; n++) 
                {
                    for (int m=max(-n,-j+k+n); m<=min(k-1,n); m++) 
                    {
                        int jnkms = (j - n) * (j - n + 1) / 2 + k - m;
                        int nm    = n * n + n - m;
                        temp_M += child_M[jnkms] * Ynm[nm] * zeta_fmm::real(ipow2n(m) * oddOrEven(n));
                    }
                    for (int m=k; m<=min(n,j+k-n); m++) 
                    {
                        int jnkms = (j - n) * (j - n + 1) / 2 - k + m;
                        int nm    = n * n + n - m;
                        temp_M += child_M[jnkms].conj() * Ynm[nm] * zeta_fmm::real(oddOrEven(k+n+m));
                    }
                }
                M[jks] += temp_M;
            }
        }
    }
}

__global__ void m2m_image(zeta_fmm::Cell3* cells, int* image_cells, int offset, int P, zeta_fmm::complex* Ms, zeta_fmm::real cycle)
{
    int cell_idx = image_cells[blockIdx.x + offset];
    int pole_size_eachcell = get_pole_size_eachcell(P);
    zeta_fmm::Cell3* cell = cells + cell_idx;
    zeta_fmm::complex* M = Ms + cell_idx * pole_size_eachcell;

    __shared__ zeta_fmm::complex Ynm[512];

    int child_cell_idx = cell->child_info.offset;
    zeta_fmm::Cell3* child_cell = cells + child_cell_idx;
    zeta_fmm::complex* child_M = Ms + child_cell_idx * pole_size_eachcell;
    
    for(int pz = -1; pz <= 1; pz++)
    {
        for(int py = -1; py <= 1; py++)
        {
            for(int px = -1; px <= 1; px++)
            {
                zeta_fmm::vec3r dx = cell->center - child_cell->center - zeta_fmm::vec3r(px,py,pz) * cycle;
                zeta_fmm::Sph sph = dx.toSph2();
                calc_Ynm(Ynm, P, sph.rho, sph.theta, sph.phi);
                for(int j = 0; j <= P; j++) 
                {
                    for (int k = 0; k <= j; k++) 
                    {
                        int jks = j * (j + 1) / 2 + k;
                        zeta_fmm::complex temp_M;
                        temp_M.rel = 0; temp_M.img = 0;
                        for (int n=0; n<=j; n++) 
                        {
                            for (int m=max(-n,-j+k+n); m<=min(k-1,n); m++) 
                            {
                                int jnkms = (j - n) * (j - n + 1) / 2 + k - m;
                                int nm    = n * n + n - m;
                                temp_M += child_M[jnkms] * Ynm[nm] * zeta_fmm::real(ipow2n(m) * oddOrEven(n));
                            }
                            for (int m=k; m<=min(n,j+k-n); m++) 
                            {
                                int jnkms = (j - n) * (j - n + 1) / 2 - k + m;
                                int nm    = n * n + n - m;
                                temp_M += child_M[jnkms].conj() * Ynm[nm] * zeta_fmm::real(oddOrEven(k+n+m));
                            }
                        }
                        M[jks] += temp_M;
                    }
                }
            }
        }
    }
}

__global__ void l2l(zeta_fmm::Cell3* cells, int* branch_cells, int offset, int P, zeta_fmm::complex* Ls)
{
    int cell_idx = branch_cells[blockIdx.x + offset];
    int pole_size_eachcell = get_pole_size_eachcell(P);
    zeta_fmm::Cell3* cell = cells + cell_idx;
    zeta_fmm::complex* L = Ls + cell_idx * pole_size_eachcell;

    __shared__ zeta_fmm::complex Ynm[512];

    for(int i = 0; i < cell->child_info.number; i++)
    {
        int child_cell_idx = cell->child_info.offset + i;
        zeta_fmm::Cell3* child_cell = cells + child_cell_idx;
        zeta_fmm::complex* child_L = Ls + child_cell_idx * pole_size_eachcell;        
        zeta_fmm::vec3r dX = child_cell->center - cell->center;
        zeta_fmm::Sph sph = dX.toSph2();
        calc_Ynm(Ynm, P, sph.rho, sph.theta, sph.phi);
        for (int j=0; j<=P; j++) 
        {
            for (int k=0; k<=j; k++) 
            {
                int jks = j * (j + 1) / 2 + k;
                zeta_fmm::complex temp_L;
                temp_L.rel = 0; temp_L.img = 0;
                for (int n=j; n<=P; n++) 
                {
                    for (int m=j+k-n; m<0; m++) 
                    {
                        int jnkm = (n - j) * (n - j) + n - j + m - k;
                        int nms  = n * (n + 1) / 2 - m;
                        temp_L += L[nms].conj() * Ynm[jnkm] * zeta_fmm::real(oddOrEven(k));
                    }
                    for (int m=0; m<=n; m++) 
                    {
                        if (n-j >= abs(m-k)) 
                        {
                            int jnkm = (n - j) * (n - j) + n - j + m - k;
                            int nms  = n * (n + 1) / 2 + m;
                            temp_L += L[nms] * Ynm[jnkm] * zeta_fmm::real(oddOrEven((m-k)*(m<k)));
                        }
                    }
                }
                child_L[jks] += temp_L;
            }
        }
    }
}

__global__ void p2p(
    zeta_fmm::Body3* bodies, 
    zeta_fmm::Cell3* cells, 
    int* p2p_matrix, zeta_fmm::Offset3rPadding* p2p_offset_matrix, int p2p_matrix_col
)
{
    int tar_cell_idx = p2p_matrix[blockIdx.x * p2p_matrix_col];
    int src_cell_num = p2p_matrix[blockIdx.x * p2p_matrix_col + 1];
    zeta_fmm::Cell3* tar_cell = cells + tar_cell_idx;

    for(int offset = 0; offset < src_cell_num; offset++)
    {
        int src_cell_idx = p2p_matrix[blockIdx.x * p2p_matrix_col + 2 + offset];
        zeta_fmm::Offset3rPadding src_cell_offset = p2p_offset_matrix[blockIdx.x * (p2p_matrix_col - 2) + offset];
        zeta_fmm::Cell3* src_cell = cells + src_cell_idx;

        for(int tar_offset = 0; tar_offset < tar_cell->body_info.number; tar_offset += P2P_KERNEL_TH_NUM)
        {
            if(tar_offset + threadIdx.x < tar_cell->body_info.number)
            {
                int tar_body_idx = tar_cell->body_info.offset + tar_offset + threadIdx.x;
                zeta_fmm::Body3* b_tar = bodies + tar_body_idx;
                zeta_fmm::real p = 0.0;
                zeta_fmm::vec3r f(0,0,0);
                for(int j = src_cell->body_info.offset; j < src_cell->body_info.offset + src_cell->body_info.number; j++)
                {
                    zeta_fmm::Body3* b_src = bodies + j;
                    zeta_fmm::vec3r dx = b_tar->loc - add(b_src->loc, src_cell_offset);
                    zeta_fmm::real r = dx.r();
                    if(r > 0)
                    {
                        p += b_src->q / r;
                        f += b_src->q / r / r * (-dx) / r;
                    }
                }
                b_tar->p += p;
                b_tar->f += f;
            }
        }
    }
}

__global__ void p2p_reg(
    zeta_fmm::Body3* bodies, 
    zeta_fmm::Cell3* cells, 
    int* p2p_matrix, zeta_fmm::Offset3rPadding* p2p_offset_matrix, int p2p_matrix_col,
    zeta_fmm::real rega
)
{
    int tar_cell_idx = p2p_matrix[blockIdx.x * p2p_matrix_col];
    int src_cell_num = p2p_matrix[blockIdx.x * p2p_matrix_col + 1];
    zeta_fmm::Cell3* tar_cell = cells + tar_cell_idx;

    for(int offset = 0; offset < src_cell_num; offset++)
    {
        int src_cell_idx = p2p_matrix[blockIdx.x * p2p_matrix_col + 2 + offset];
        zeta_fmm::Offset3rPadding src_cell_offset = p2p_offset_matrix[blockIdx.x * (p2p_matrix_col - 2) + offset];
        zeta_fmm::Cell3* src_cell = cells + src_cell_idx;

        for(int tar_offset = 0; tar_offset < tar_cell->body_info.number; tar_offset += P2P_KERNEL_TH_NUM)
        {
            if(tar_offset + threadIdx.x < tar_cell->body_info.number)
            {
                int tar_body_idx = tar_cell->body_info.offset + tar_offset + threadIdx.x;
                zeta_fmm::Body3* b_tar = bodies + tar_body_idx;
                zeta_fmm::real wi = get_w(b_tar->loc - tar_cell->center, tar_cell->r, rega);
                #if FMM_REG_USE_DW
                zeta_fmm::vec3r dwi = get_dw(b_tar->loc - tar_cell->center, tar_cell->r, rega);
                #endif
                zeta_fmm::real p = 0.0;
                zeta_fmm::vec3r f(0,0,0);
                for(int j = src_cell->body_info.offset; j < src_cell->body_info.offset + src_cell->body_info.number; j++)
                {
                    zeta_fmm::Body3* b_src = bodies + j;
                    zeta_fmm::real wj = get_w(b_src->loc - src_cell->center, src_cell->r, rega);
                    zeta_fmm::vec3r dx = b_tar->loc - add(b_src->loc, src_cell_offset);
                    zeta_fmm::real r = dx.r();
                    if(r > 0)
                    {
                        p += wj * b_src->q / r;
                        f += wj * b_src->q / r / r * (-dx) / r;
                    }
                }
                b_tar->p += wi * p;
                #if FMM_REG_USE_DW
                b_tar->f += wi * f + dwi * p;
                #else
                b_tar->f += wi * f;
                #endif
            }
        }
    }
}

__global__ void m2l(
    zeta_fmm::Cell3* cells, 
    int* m2l_matrix, zeta_fmm::Offset3rPadding* m2l_offset_matrix, int m2l_matrix_col, 
    int P, zeta_fmm::complex* Ms, zeta_fmm::complex* Ls
)
{
    int tar_cell_idx = m2l_matrix[blockIdx.x * m2l_matrix_col];
    int src_cell_num = m2l_matrix[blockIdx.x * m2l_matrix_col + 1];
    int pole_size_eachcell = get_pole_size_eachcell(P);

    __shared__ zeta_fmm::complex Ynm[512];
    
    zeta_fmm::Cell3* tar_cell = cells + tar_cell_idx;
    zeta_fmm::complex* tar_L = Ls + tar_cell_idx * pole_size_eachcell;
    
    for(int i = 0; i < src_cell_num; i++)
    {
        int src_cell_idx = m2l_matrix[blockIdx.x * m2l_matrix_col + 2 + i];
        zeta_fmm::Offset3rPadding src_cell_offset = m2l_offset_matrix[blockIdx.x * (m2l_matrix_col - 2) + i];
        zeta_fmm::Cell3* src_cell = cells + src_cell_idx;
        zeta_fmm::complex* src_M = Ms + src_cell_idx * pole_size_eachcell;
        
        zeta_fmm::vec3r dX = tar_cell->center - add(src_cell->center, src_cell_offset);
        zeta_fmm::Sph sph = dX.toSph2();
        calc_Ynm2(Ynm, P, sph.rho, sph.theta, sph.phi);
        for (int j=0; j<=P; j++) 
        {
            zeta_fmm::real Cnm = oddOrEven(j);
            for (int k=0; k<=j; k++) 
            {
                int jks = j * (j + 1) / 2 + k;
                zeta_fmm::complex temp_L;
                temp_L.rel = 0; temp_L.img = 0;
                for (int n=0; n<=P; n++) 
                {
                    if(j + n <= P)   // WARNING : this line is forgot in exafmm/minial(if lack, jnkm for Ynm will be out of bounds)
                    {
                        for (int m=-n; m<0; m++) 
                        {
                            int nms  = n * (n + 1) / 2 - m;
                            int jnkm = (j + n) * (j + n) + j + n + m - k;
                            temp_L += src_M[nms].conj() * Cnm * Ynm[jnkm];
                        }
                        for (int m=0; m<=n; m++) 
                        {
                            int nms  = n * (n + 1) / 2 + m;
                            int jnkm = (j + n) * (j + n) + j + n + m - k;
                            zeta_fmm::real Cnm2 = Cnm * oddOrEven((k-m)*(k<m)+m);
                            temp_L += src_M[nms] * Cnm2 * Ynm[jnkm];
                        }
                    }
                }
                tar_L[jks] += temp_L;
            }
        }
    }
}

__global__ void l2p(
    zeta_fmm::Body3* bodies, 
    zeta_fmm::Cell3* cells, 
    int* leaf_cells, 
    int P, 
    zeta_fmm::complex* Ls
)
{
    int cell_idx = leaf_cells[blockIdx.x];
    int pole_size_eachcell = get_pole_size_eachcell(P);
    zeta_fmm::Cell3* cell = cells + cell_idx;
    zeta_fmm::complex* L = Ls + cell_idx * pole_size_eachcell;

    __shared__ zeta_fmm::complex Ynm[512];
    __shared__ zeta_fmm::complex YnmD[512];

    for(int i = 0; i < cell->body_info.number; i++)
    {
        zeta_fmm::Body3* body = bodies + cell->body_info.offset + i;
        zeta_fmm::vec3r dx = body->loc - cell->center;
        zeta_fmm::Sph sph = dx.toSph2();
        calc_Ynm(Ynm, P, sph.rho, sph.theta, sph.phi);
        calc_YnmD(YnmD, P, sph.rho, sph.theta, sph.phi);
        zeta_fmm::vec3r temp = {0,0,0};
        zeta_fmm::complex I;
        I.rel = 0; I.img = 1;
        zeta_fmm::real p = 0;
        for (int n=0; n<=P; n++)
        {
            int nm  = n * n + n;
            int nms = n * (n + 1) / 2;
            p += (L[nms] * Ynm[nm]).rel;
            temp[0] += (L[nms] * Ynm[nm]).rel / sph.rho * n;
            temp[1] += (L[nms] * YnmD[nm]).rel;
            for (int m=1; m<=n; m++) 
            {
                nm  = n * n + n + m;
                nms = n * (n + 1) / 2 + m;
                p += 2 * (L[nms] * Ynm[nm]).rel;
                temp[0] += 2 * (L[nms] * Ynm[nm]).rel / sph.rho * n;
                temp[1] += 2 * (L[nms] * YnmD[nm]).rel;
                temp[2] += 2 * (L[nms] * Ynm[nm] * I).rel * m;
            }
        }
        body->p += p;
        zeta_fmm::vec3r cart = derivate_sph2cart(sph, temp);
        body->f += cart;
    }
}

__global__ void l2p_dummy(
    zeta_fmm::Body3* bodies, 
    zeta_fmm::Cell3* cells, 
    int* leaf_cells, 
    int P, 
    zeta_fmm::complex* Ls
)
{
    int cell_idx = leaf_cells[blockIdx.x];
    int pole_size_eachcell = get_pole_size_eachcell(P);
    zeta_fmm::Cell3* cell = cells + cell_idx;
    zeta_fmm::complex* L = Ls + cell_idx * pole_size_eachcell;

    __shared__ zeta_fmm::complex Ynm[512];
    __shared__ zeta_fmm::complex YnmD[512];

    for(int i = 0; i < cell->body_info.number; i++)
    {
        zeta_fmm::Body3* body = bodies + cell->body_info.offset + i;
        zeta_fmm::vec3r dx = body->loc - cell->center;
        zeta_fmm::Sph sph = dx.toSph2();
        calc_Ynm(Ynm, P, sph.rho, sph.theta, sph.phi);
        calc_YnmD(YnmD, P, sph.rho, sph.theta, sph.phi);
        zeta_fmm::vec3r temp = {0,0,0};
        zeta_fmm::complex I;
        I.rel = 0; I.img = 1;
        zeta_fmm::real p = 0;
        for (int n=0; n<=P; n++)
        {
            int nm  = n * n + n;
            int nms = n * (n + 1) / 2;
            p += (L[nms] * Ynm[nm]).rel;
            temp[0] += (L[nms] * Ynm[nm]).rel / sph.rho * n;
            temp[1] += (L[nms] * YnmD[nm]).rel;
            for (int m=1; m<=n; m++) 
            {
                nm  = n * n + n + m;
                nms = n * (n + 1) / 2 + m;
                p += 2 * (L[nms] * Ynm[nm]).rel;
                temp[0] += 2 * (L[nms] * Ynm[nm]).rel / sph.rho * n;
                temp[1] += 2 * (L[nms] * YnmD[nm]).rel;
                temp[2] += 2 * (L[nms] * Ynm[nm] * I).rel * m;
            }
        }
        body->p += p;
        zeta_fmm::vec3r cart = derivate_sph2cart(sph, temp);
        body->f += cart;
    }
}

__global__ void l2p_reg(
    zeta_fmm::Body3* bodies,
    zeta_fmm::Cell3* cells, 
    int* leaf_cells, 
    int P, 
    zeta_fmm::complex* Ls, 
    zeta_fmm::real rega
)
{
    int cell_idx = leaf_cells[blockIdx.x];
    int pole_size_eachcell = get_pole_size_eachcell(P);
    zeta_fmm::Cell3 cell = cells[cell_idx];
    zeta_fmm::complex* L = Ls + cell_idx * pole_size_eachcell;

    __shared__ zeta_fmm::complex Ynm[512];
    __shared__ zeta_fmm::complex YnmD[512];

    for(int i = 0; i < cell.body_info.number; i++)
    {
        int bidx = cell.body_info.offset + i;
        zeta_fmm::Body3* body = bodies + bidx;
        zeta_fmm::vec3r dx = body->loc - cell.center;
        zeta_fmm::real w = get_w(dx, cell.r, rega);
        #ifdef FMM_REG_USE_DW
        zeta_fmm::vec3r dw = get_dw(dx, cell.r, rega);
        #endif
        zeta_fmm::Sph sph = dx.toSph2();
        calc_Ynm(Ynm, P, sph.rho, sph.theta, sph.phi);
        calc_YnmD(YnmD, P, sph.rho, sph.theta, sph.phi);
        zeta_fmm::vec3r temp = {0,0,0};
        zeta_fmm::complex I;
        I.rel = 0; I.img = 1;
        zeta_fmm::real p = 0;
        for (int n=0; n<=P; n++)
        {
            int nm  = n * n + n;
            int nms = n * (n + 1) / 2;
            p += (L[nms] * Ynm[nm]).rel;
            temp[0] += (L[nms] * Ynm[nm]).rel / sph.rho * n;
            temp[1] += (L[nms] * YnmD[nm]).rel;
            for (int m=1; m<=n; m++) 
            {
                nm  = n * n + n + m;
                nms = n * (n + 1) / 2 + m;
                p += 2 * (L[nms] * Ynm[nm]).rel;
                temp[0] += 2 * (L[nms] * Ynm[nm]).rel / sph.rho * n;
                temp[1] += 2 * (L[nms] * YnmD[nm]).rel;
                temp[2] += 2 * (L[nms] * Ynm[nm] * I).rel * m;
            }
        }
        zeta_fmm::vec3r cart = derivate_sph2cart(sph, temp);
        #if FMM_REG_USE_DW
        body->f += w * cart + p * dw;
        #else
        body->f += w * cart;
        #endif
        body->p += w * p;
    }
}

__global__ void arrange_bodies(
    zeta_fmm::Body3* bodies, 
    zeta_fmm::Cell3* cells, 
    int* leaf_cells, 
    int merged_num_body, zeta_fmm::real* premerged_body_p, zeta_fmm::vec3r* premerged_body_f
)
{
    int cell_idx = leaf_cells[blockIdx.x];
    zeta_fmm::Cell3 cell = cells[cell_idx];

    for(int i = 0; i < cell.body_info.number; i++)
    {
        int bidx = cell.body_info.offset + i;
        zeta_fmm::Body3 body = bodies[bidx];
        int partion_idx = ((body.loc[0] > cell.center[0]) << 2) 
                        + ((body.loc[1] > cell.center[1]) << 1) 
                        + ((body.loc[2] > cell.center[2]) << 0);
        premerged_body_p[merged_num_body * partion_idx + body.idx] = body.p;
        premerged_body_f[merged_num_body * partion_idx + body.idx] = body.f;
    }
}

__global__ void merge_bodies(
    zeta_fmm::Body3* merged_bodies, 
    int merged_num_body, zeta_fmm::real* premerged_body_p, zeta_fmm::vec3r* premerged_body_f
)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < merged_num_body)
    {
        zeta_fmm::real p = 0;
        zeta_fmm::vec3r f(0,0,0);
        for(int i = 0; i < 8; i++)
        {
            p += premerged_body_p[merged_num_body * i + idx];
            f += premerged_body_f[merged_num_body * i + idx];
        }
        merged_bodies[idx].p = p;
        merged_bodies[idx].f = f;
    }
}

zeta_fmm::GPUKernel::GPUKernel()
{
    //std::cout<<"init GPUKernel"<<std::endl;
}

void zeta_fmm::GPUKernel::solve(
    int P, zeta_fmm::real rega,
    Body3* bodies, int real_num_body, 
    Body3* merged_bodies, int merged_num_body,
    Cell3* cells, int num_cell, 
    zeta_fmm::complex* Ms, zeta_fmm::complex* Ls,
    int* leaf_cells, int leaf_cell_num,
    int* branch_cells, std::vector<zeta_fmm::OffsetAndNumber> level_infos,
    int* image_cells, int image_cell_num, zeta_fmm::real cycle,
    int* p2p_matrix, Offset3rPadding* p2p_offset_matrix, int p2p_matrix_rows, int p2p_matrix_col,
    int* m2l_matrix, Offset3rPadding* m2l_offset_matrix, int m2l_matrix_rows, int m2l_matrix_col,
    int dummy
)
{
    // P2M
    if(!dummy)
    {
        if(rega == 0)
            p2m<<<leaf_cell_num, 1>>>(bodies, cells, leaf_cells, P, Ms);
        else
            p2m_reg<<<leaf_cell_num, 1>>>(bodies, cells, leaf_cells, P, Ms, rega);
    }
    else
    {
        p2m_dummy<<<leaf_cell_num, 1>>>(bodies, cells, leaf_cells, P, Ms);
        int pole_size_eachcell = get_pole_size_eachcell(P);
        zeta_fmm::complex* c_Ls = new zeta_fmm::complex[num_cell * pole_size_eachcell];
        hipMemcpy(c_Ls, Ms, num_cell * pole_size_eachcell * sizeof(zeta_fmm::complex), hipMemcpyDeviceToHost);
        for(int i = 0; i < num_cell; i++)
        {
            printf("M[%d]= ", i);
            for(int p = 0; p < pole_size_eachcell; p++)
            {
                zeta_fmm::complex L = c_Ls[i * pole_size_eachcell + p];
                printf("(%.8f,%.8f)  ", L.rel, L.img);
            }
            printf("\n");
        }
        delete[] c_Ls;
    }

    // M2M
    for(int i = 0; i < level_infos.size(); i++)
    {
        zeta_fmm::OffsetAndNumber offset_number = level_infos[i];
        m2m<<<offset_number.number, 1>>>(cells, branch_cells, offset_number.offset, P, Ms);
    }

    // M2M for image cells
    zeta_fmm::real c = cycle;
    for(int i = 0; i < image_cell_num - 1; i++) // the last one is not necessary
    {
        m2m_image<<<1,1>>>(cells, image_cells, i, P, Ms, c);
        c *= 3;
    }

    // M2L
    m2l<<<m2l_matrix_rows,1>>>(cells, m2l_matrix, m2l_offset_matrix, m2l_matrix_col, P, Ms, Ls);

    // P2P  
    if(rega == 0)
    {
        p2p<<<p2p_matrix_rows,P2P_KERNEL_TH_NUM>>>(bodies, cells, p2p_matrix, p2p_offset_matrix, p2p_matrix_col);
    }
    else
    {
        p2p_reg<<<p2p_matrix_rows,P2P_KERNEL_TH_NUM>>>(bodies, cells, p2p_matrix, p2p_offset_matrix, p2p_matrix_col, rega);
    }

    // L2L
    for(int i = level_infos.size() - 1; i >= 0; i--)
    {
        zeta_fmm::OffsetAndNumber offset_number = level_infos[i];
        l2l<<<offset_number.number, 1>>>(cells, branch_cells, offset_number.offset, P, Ls);
    }

    // L2P
    
    if(rega == 0)
        l2p<<<leaf_cell_num, 1>>>(bodies, cells, leaf_cells, P, Ls);
    else
        l2p_reg<<<leaf_cell_num, 1>>>(bodies, cells, leaf_cells, P, Ls, rega);

    // arrange and merge reg bodies
    if(rega > 0)
    {
        zeta_fmm::real* g_premerged_body_p;
        hipMalloc(&g_premerged_body_p, merged_num_body * 8 * sizeof(zeta_fmm::real));
        hipMemset(g_premerged_body_p, 0.0, merged_num_body * 8 * sizeof(zeta_fmm::real));
        zeta_fmm::vec3r* g_premerged_body_f;
        hipMalloc(&g_premerged_body_f, merged_num_body * 8 * sizeof(zeta_fmm::vec3r));
        hipMemset(g_premerged_body_f, 0.0, merged_num_body * 8 * sizeof(zeta_fmm::vec3r));

        arrange_bodies<<<leaf_cell_num, 1>>>(bodies, cells, leaf_cells, merged_num_body, g_premerged_body_p, g_premerged_body_f);

        int merge_block_num = (merged_num_body + 1023) / 1024;
        merge_bodies<<<merge_block_num, 1024>>>(merged_bodies, merged_num_body, g_premerged_body_p, g_premerged_body_f);

        hipFree(g_premerged_body_p);
        hipFree(g_premerged_body_f);
    }
}